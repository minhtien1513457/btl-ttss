#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <stdbool.h>
#include <math.h>
#include <iostream>
using namespace std;

struct node
{
	int data;
	struct node *next;
};

struct Itemset
{
	string data;
	struct Itemset *next;
};

struct ItemsetCollection
{
	Itemset *data;
	struct ItemsetCollection *next;
};

struct rules
{
	Itemset *X;
	Itemset *Y;
	double sp;
	double cf;
	struct rules *next;
};

//hien thi danh sach
void print_node(node *head)
{
	struct node *ptr = head;
	if (ptr == NULL) cout << "this object is empty";


	cout << "\n[ ";

	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		cout << ptr->data;
		ptr = ptr->next;
	}

	cout << " ]";
}
void print_ItemsetRules(Itemset *head)
{
	struct Itemset *ptr = head;
	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		cout << ptr->data;
		ptr = ptr->next;
	}
}
void print_Itemset(Itemset *head)
{
	struct Itemset *ptr = head;
	if (ptr == NULL) cout << "\n this object is empty";


	cout << "\n[ ";

	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		cout << "(" << ptr->data << ") ";
		ptr = ptr->next;
	}

	cout << " ]";
}
void print_ItemsetCollection(ItemsetCollection *head)
{
	struct ItemsetCollection *ptr = head;
	cout << "\n[ ";

	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		print_Itemset(ptr->data);
		ptr = ptr->next;
	}

	cout << "\n]";
}
void print_rules(rules *head)
{
	struct rules *ptr = head;
	cout << "\n[\n ";

	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		print_ItemsetRules(ptr->X);
		cout << " => ";
		print_ItemsetRules(ptr->Y);
		cout << " " << ptr->sp << " ";
		cout << ptr->cf;
		cout << "\n";
		ptr = ptr->next;
	}

	cout << "]";
}

//chen link tai vi tri dau tien
void insertFirst_node(node *&head, int data)
{
	//tao mot link
	struct node *link = (struct node*) malloc(sizeof(struct node));

	link->data = data;

	//tro link nay toi first Itemset cu
	link->next = head;

	//tro first toi first Itemset moi
	head = link;
}
void insertFirst_Itemset(Itemset *&head, string data)
{
	//tao mot link
	Itemset *link = new Itemset();

	link->data = data;

	//tro link nay toi first Itemset cu
	link->next = head;

	//tro first toi first Itemset moi
	head = link;
}
void insertFirst_rules(rules *&head, Itemset *X, Itemset *Y, double sp, double cf)
{
	//tao mot link
	struct rules *link = (struct rules*) malloc(sizeof(struct rules));

	link->X = X;
	link->Y = Y;
	link->cf = cf;
	link->sp = sp;

	//tro link nay toi first Itemset cu
	link->next = head;

	//tro first toi first Itemset moi
	head = link;
}
void insertFirst_ItemsetCollection_cuda(ItemsetCollection *&head, Itemset *data)
{
	//tao mot link
	struct ItemsetCollection *link = (struct ItemsetCollection*) malloc(sizeof(struct ItemsetCollection));

	link->data = data;

	//tro link nay toi first Itemset cu
	link->next = head;

	//tro first toi first Itemset moi
	head = link;
}
 void insertFirst_ItemsetCollection(ItemsetCollection *&head, Itemset *data)
{
	//tao mot link
	struct ItemsetCollection *link = (struct ItemsetCollection*) malloc(sizeof(struct ItemsetCollection));

	link->data = data;

	//tro link nay toi first Itemset cu
	link->next = head;

	//tro first toi first Itemset moi
	head = link;
}
//xoa phan tu dau tien
struct Itemset* deleteFirst(Itemset *head)
{

	//luu tham chieu toi first link
	struct Itemset *tempLink = head;

	//danh dau next toi first link la first 
	head = head->next;

	//tra ve link bi xoa
	return tempLink;
}

//kiem tra list co trong hay khong
bool isEmpty(Itemset *head)
{
	return head == NULL;
}

int length_Itemset(Itemset *head)
{
	int length = 0;
	struct Itemset *current;

	for (current = head; current != NULL; current = current->next)
	{
		length++;
	}

	return length;
}
int length_node(node *head)
{
	int length = 0;
	struct node *current;

	for (current = head; current != NULL; current = current->next)
	{
		length++;
	}

	return length;
}
int length_ItemsetCollection(ItemsetCollection *head)
{
	int length = 0;
	struct ItemsetCollection *current;

	for (current = head; current != NULL; current = current->next)
	{
		length++;
	}

	return length;
}
//tim mot link voi key da cho
struct Itemset* find_(Itemset *head, string data) {

	//bat dau tim tu first link
	struct Itemset* current = head;

	//neu list la trong
	if (head == NULL)
	{
		return NULL;
	}

	//duyet qua list
	while (current->data != data) {

		//neu day la last Itemset
		if (current->next == NULL) {
			return NULL;
		}
		else {
			//di chuyen toi next link
			current = current->next;
		}
	}

	//neu tim thay du lieu, tra ve link hien tai
	return current;
}

//tim gia tri cua node tai vi tri index
int findData_node(node *head, int index)
{
	int dem = 0;
	//bat dau tim tu first link
	struct node* current = head;

	//neu list la trong
	if (head == NULL)
	{
		return NULL;
	}

	//duyet qua list
	while (dem != index)
	{
		//di chuyen toi next link
		current = current->next;
		dem++;
	}

	//neu tim thay du lieu, tra ve link hien tai
	return current->data;

}

//tim gia tri cua itemset tai vi tri index
string findData_Itemset(Itemset *head, int index)
{
	int dem = 0;
	//bat dau tim tu first link
	struct Itemset* current = head;

	//neu list la trong
	if (head == NULL)
	{
		return "";
	}

	//duyet qua list
	while (dem != index)
	{
		//di chuyen toi next link
		current = current->next;
		dem++;
	}

	//neu tim thay du lieu, tra ve link hien tai
	return current->data;

}

//xoa mot link voi key da cho
struct Itemset* deleteKey(Itemset *&head, string data) {

	//bat dau tu first link
	struct Itemset* current = head;
	struct Itemset* previous = NULL;

	//neu list la trong
	if (head == NULL) {
		return NULL;
	}

	//duyet qua list
	while (current->data != data) {

		//neu day la last Itemset
		if (current->next == NULL) {
			return NULL;
		}
		else {
			//luu tham chieu toi link hien tai
			previous = current;
			//di chuyen toi next link
			current = current->next;
		}

	}

	//cap nhat link
	if (current == head) {
		//thay doi first de tro toi next link
		//head = NULL;
		head = head->next;
	}
	else {
		//bo qua link hien tai
		previous->next = current->next;
		return current;
	}


}

// ham sap xep
void sort(Itemset *head) {

	int i, j, k, tempKey;
	string tempData;
	struct Itemset *current;
	struct Itemset *next;

	int size = length_Itemset(head);
	k = size;

	for (i = 0; i < size - 1; i++, k--) {
		current = head;
		next = head->next;

		for (j = 1; j < k; j++) {

			if (current->data > next->data) {
				tempData = current->data;
				current->data = next->data;
				next->data = tempData;
			}

			current = current->next;
			next = next->next;
		}
	}
}

// ham dao nguoc list
void reverse_Itemset(struct Itemset** head_ref) {
	struct Itemset* prev = NULL;
	struct Itemset* current = *head_ref;
	struct Itemset* next;

	while (current != NULL) {
		next = current->next;
		current->next = prev;
		prev = current;
		current = next;
	}

	*head_ref = prev;
}
void reverse_ItemsetCollection(struct ItemsetCollection** head_ref) {
	struct ItemsetCollection* prev = NULL;
	struct ItemsetCollection* current = *head_ref;
	struct ItemsetCollection* next;

	while (current != NULL) {
		next = current->next;
		current->next = prev;
		prev = current;
		current = next;
	}

	*head_ref = prev;
}

////////////////////////////////////////////////////////////////////////////////
//******************************************************************************

//clear all ItemsetCollection
void clearItemsetCollection(ItemsetCollection *&head) {
	//bat dau tu phan dau danh sach
	while (head != NULL)
	{
		head = head->next;
	}
}

//count ItemsetCollection
int countItemsetCollection(ItemsetCollection *head) {
	//bat dau tu phan dau danh sach
	int kq = 0;
	while (head != NULL)
	{
		kq++;
		head = head->next;
	}
	return kq;
}
int countRules(rules *head) {
	//bat dau tu phan dau danh sach
	int kq = 0;
	while (head != NULL)
	{
		kq++;
		head = head->next;
	}
	return kq;
}
//count Itemset
int countItemset(Itemset *head) {
	//bat dau tu phan dau danh sach
	int kq = 0;
	while (head != NULL)
	{
		kq++;
		head = head->next;
	}
	return kq;
}

//ttim so lan xuat hien itemset trong itemcollection
int Appear(ItemsetCollection *db, Itemset *item) {
	int db_size = countItemsetCollection(db);
	int item_size = countItemset(item);
	struct ItemsetCollection *ptr_1 = db;
	int dem_z = 0;
	while (ptr_1 != NULL)
	{
		int dem = 0;
		struct Itemset *tmp = item;
		while (tmp != NULL)
		{
			if (find_(ptr_1->data, tmp->data) != NULL)
			{
				dem++;
			}
			tmp = tmp->next;
			if (dem == item_size) {
				dem_z++;
			}
		}

		ptr_1 = ptr_1->next;
	}
	return dem_z;

}

//ham tinh do pho bien
double FindSupport(ItemsetCollection *db, Itemset *item) {
	double kq = 0.0;
	kq = ((double)Appear(db, item) / (double)countItemsetCollection(db)) * 100;
	return kq;
}

int GetBit(int value, int position)
{
	int bit = value & (int)pow(2.0, position);
	return (bit > 0 ? 1 : 0);
}
//doi thap phan sang nhi phan
node* DecimalToBinary(int value, int length)
{
	struct node *binary = NULL;
	for (int position = 0; position < length; position++)
	{
		insertFirst_node(binary, GetBit(value, position));
	}
	return (binary);
}

//dem so bit 1 trong chuoi nhi phan
int GetOnCount(int value, int length)
{
	int dem = 0;
	node* binary = DecimalToBinary(value, length);
	struct node *ptr = binary;
	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		if (ptr->data == 1) dem++;
		ptr = ptr->next;
	}
	return dem;
}

//ham tim tat ca tap con cua tap co k phan tu
struct ItemsetCollection* FindSubsets(Itemset *itemset, int n)
{
	ItemsetCollection *subsets = NULL;
	int subsetCount = (int)pow(2.0, countItemset(itemset));

	for (int i = 0; i < subsetCount; i++)
	{
		if (n == 0 || GetOnCount(i, countItemset(itemset)) == n)
		{
			node* binary = DecimalToBinary(i, countItemset(itemset));

			Itemset *subset = NULL;
			for (int nodeIndex = 0; nodeIndex < length_node(binary); nodeIndex++)
			{
				if (findData_node(binary, nodeIndex) == 1)
				{
					insertFirst_Itemset(subset, findData_Itemset(itemset, nodeIndex));
				}
			}
			insertFirst_ItemsetCollection(subsets, subset);
		}
	}
	return subsets;
}
//--------------------------------mine data with sp----------------------------------------------------------//
struct Itemset* GetUniqueItems(ItemsetCollection *head) {
	Itemset *kq = NULL;
	struct ItemsetCollection *ptr_1 = head;

	insertFirst_Itemset(kq, ptr_1->data->data);
	while (ptr_1 != NULL)
	{
		Itemset *ptr_2 = ptr_1->data;
		while (ptr_2 != NULL)
		{
			if (find_(kq, ptr_2->data) == NULL)
			{
				insertFirst_Itemset(kq, ptr_2->data);
			}
			ptr_2 = ptr_2->next;

		}
		ptr_1 = ptr_1->next;
	}
	return kq;
}
struct ItemsetCollection* doApriori(ItemsetCollection *db, double supportThreshold,int first[]) {
	Itemset *I = GetUniqueItems(db);
	ItemsetCollection *L = NULL;//  tap du lieu pho bien
	ItemsetCollection *Li = NULL;// tap du lieu 
	ItemsetCollection *Ci = NULL;// tap du lieu duoc luot bot
	//duyet su lap lai cua phan tu dau tien trong tap du lieu
	while (I != NULL)
	{
		Itemset *tmp = NULL;
		insertFirst_Itemset(tmp, I->data);
		insertFirst_ItemsetCollection(Ci, tmp);
		I = I->next;
	}
	//
	int first_tt = length_ItemsetCollection(Ci)-1;
	ItemsetCollection *Ci_tmp1 = Ci;
	while (Ci_tmp1 != NULL)
	{
		if (first[first_tt] >= supportThreshold)
		{
			insertFirst_ItemsetCollection(Li, Ci_tmp1->data);
			insertFirst_ItemsetCollection(L, Ci_tmp1->data);
		}
		first_tt--;
		Ci_tmp1 = Ci_tmp1->next;
	}
	clearItemsetCollection(Ci);
	Ci = FindSubsets(GetUniqueItems(Li), 2);
	int k = 3;
	//
	//su lap lai cac lan ke tiep
	for (int i = 0; i < length_ItemsetCollection(Ci); i++)
	{
		//lay Li tu Ci (phan tu dc luot bo)
		clearItemsetCollection(Li);
		ItemsetCollection *Ci_tmp = Ci;
		while (Ci_tmp != NULL)
		{
			double sp = FindSupport(db, Ci_tmp->data);
			if (sp >= supportThreshold)
			{
				insertFirst_ItemsetCollection(Li, Ci_tmp->data);
				insertFirst_ItemsetCollection(L, Ci_tmp->data);
			}
			Ci_tmp = Ci_tmp->next;
		}
		if (Li == NULL) break;
		clearItemsetCollection(Ci);
		Ci = FindSubsets(GetUniqueItems(Li), k);
		k++;
	}

	return (L);
}
void print_ItemsetCollection_sp(ItemsetCollection *head, ItemsetCollection *db)
{
	struct ItemsetCollection *ptr = head;
	cout << "\n[ ";

	//bat dau tu phan dau danh sach
	while (ptr != NULL)
	{
		print_Itemset(ptr->data);
		cout << FindSupport(db, ptr->data);
		ptr = ptr->next;
	}

	cout << "\n]";
}
//--------------------------------mine data with cf---------------------------------------------------------//
void Mine(ItemsetCollection *db, ItemsetCollection *L, int size_L, double confidenceThreshold, rules *&allRules)
{
	
	ItemsetCollection *tmp_L = L;
	for (int i = 0; i < size_L; i++)
	{
		ItemsetCollection *subsets = FindSubsets(tmp_L[i].data, 0);
		ItemsetCollection *tmp_subset = subsets;

		while (tmp_subset != NULL)
		{
			//tao ra ban copy du lieu cua tmp_L de dung cho viec xoa
			ItemsetCollection *copy_L = NULL;
			ItemsetCollection *tmp_L_L = tmp_L;
			for (int j = 0; j < size_L; j++) {
				Itemset *tmp_L_L_sub = tmp_L_L[i].data;
				Itemset *chil = NULL;
				while (tmp_L_L_sub != NULL)
				{
					insertFirst_Itemset(chil, tmp_L_L_sub->data);
					tmp_L_L_sub = tmp_L_L_sub->next;
				}
				insertFirst_ItemsetCollection(copy_L, chil);
				//tmp_L_L = tmp_L_L->next;
			}
			reverse_ItemsetCollection(&copy_L);
			//

			double confidence = (FindSupport(db, tmp_L[i].data) / FindSupport(db, tmp_subset->data))*100.0;
			if (confidence >= confidenceThreshold)
			{
				rules *rule = NULL;
				//rule->X = tmp->data;
				Itemset *tmp_X = NULL;
				tmp_X = tmp_subset->data;
				//xoa x ra khoi tmp_L->data
				while (tmp_X != NULL)
				{
					deleteKey(copy_L->data, tmp_X->data);
					tmp_X = tmp_X->next;
				}
				//rule->sp = FindSupport(db, tmp_L->data);
				//rule->cf = confidence;
				if (length_Itemset(tmp_subset->data) > 0 && length_Itemset(copy_L->data) > 0)
				{
					insertFirst_rules(allRules, tmp_subset->data, copy_L->data, FindSupport(db, tmp_L[i].data), confidence);
				}
			}
			tmp_subset = tmp_subset->next;
		}
		//tmp_L = tmp_L->next;
	}

	
}
int unique_count(ItemsetCollection *db, string a) {
	ItemsetCollection *tmp1 = db;
	int kq = 0;
	while (tmp1 != NULL) {
		Itemset *tmp2 = tmp1->data;
		while (tmp2 != NULL) {
			if (tmp2->data == a) {
				kq++;
			}
			tmp2 = tmp2->next;
		}
		tmp1 = tmp1->next;
	}
	return kq;
}
__global__ void arradd(int* a, int* b, int* c, int size)
{
	int myid = threadIdx.x;

	c[myid] = a[myid] + b[myid];
}
__global__ void additem(int *sp_dv, int *kq_dv, int size)
{
	int myid = threadIdx.x;

	kq_dv[myid] = (int)((sp_dv[myid]*100)/size);
	//kq_dv[myid] = sp_dv[myid];
}

int main() {
	struct Itemset *tmp = NULL;
	int *sp_dv;
	int *kq_dv;
	int *kq_host=new int[100];
	

	struct Itemset *a = NULL;
	struct Itemset *b = NULL;
	struct Itemset *c = NULL;
	struct Itemset *d = NULL;
	struct Itemset *e = NULL;
	struct Itemset *z = NULL;
	struct Itemset *z1 = NULL;

	insertFirst_Itemset(a, "beer");
	insertFirst_Itemset(a, "diaper");
	insertFirst_Itemset(a, "baby powder");
	insertFirst_Itemset(a, "bread");
	insertFirst_Itemset(a, "umbrella");
	print_Itemset(a);
	cout << "\n";



	insertFirst_Itemset(b, "diaper");
	insertFirst_Itemset(b, "baby powder");
	print_Itemset(b);
	cout << "\n";

	insertFirst_Itemset(c, "beer");
	insertFirst_Itemset(c, "diaper");
	insertFirst_Itemset(c, "milk");
	print_Itemset(c);
	cout << "\n";

	insertFirst_Itemset(d, "diaper");
	insertFirst_Itemset(d, "beer");
	insertFirst_Itemset(d, "detergent");
	print_Itemset(d);
	cout << "\n";

	insertFirst_Itemset(e, "beer");
	insertFirst_Itemset(e, "milk");
	insertFirst_Itemset(e, "coca-cola");
	print_Itemset(e);
	cout << "\n";

	struct ItemsetCollection *L = NULL;
	Itemset *Z_[5];
	Z_[0] = a;
	Z_[1] = b;
	Z_[2] = c;
	Z_[3] = d;
	Z_[4] = e;

	for (int i = 0; i < 5; i++) {
		insertFirst_ItemsetCollection(L, Z_[i]);
	}
	print_ItemsetCollection(L);
	cout << "\n";

	cout << "\nunique item: ";
	Itemset *uniqueItems = GetUniqueItems(L);
	print_Itemset(uniqueItems);
	cout << "\n";
	int *sp_first = new int[100];
	Itemset *uni_tmp = uniqueItems;
	int tt = 0;
	while (uni_tmp != NULL) {
		sp_first[tt] = unique_count(L, uni_tmp->data);
		tt++;
		kq_host[tt] = 0;
		uni_tmp = uni_tmp->next;
	}
	hipMalloc(&sp_dv, length_Itemset(uniqueItems) * sizeof(int));
	hipMemcpy(sp_dv, sp_first, length_Itemset(uniqueItems) * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&kq_dv, length_Itemset(uniqueItems) * sizeof(int));

	additem << < 1, length_Itemset(uniqueItems) >> > (sp_dv, kq_dv, length_ItemsetCollection(L));

	hipMemcpy(kq_host, kq_dv, length_Itemset(uniqueItems) * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(sp_dv);
	hipFree(kq_dv);
	for (int i = 0; i < 8; i++) {
		cout << " " << kq_host[i];
	}

	//covert L to array struct Itemcollection
	ItemsetCollection db[5];
	ItemsetCollection *tmp_L = L;
	for (int i = 0; i < length_ItemsetCollection(L); i++) {
		db[i].data = tmp_L->data;
		tmp_L = tmp_L->next;
	}


	//test apriori(do pho bien)
	ItemsetCollection *L1 = doApriori(L, 40.0,kq_host);
	cout << "\n itemsets in L \n" << countItemsetCollection(L1);//dem tap du lieu pho bien
	print_ItemsetCollection_sp(L1, L);

	//covert L1 to array struct Itemcollection
	ItemsetCollection db1[7];
	ItemsetCollection *tmp_L1 = L1;
	int size_L1 = length_ItemsetCollection(L1);
	for (int i = 0; i < size_L1; i++) {
		db1[i].data = tmp_L1->data;
		tmp_L1 = tmp_L1->next;
	}


	//test mining(tim luat co do tin cay >=70%)
	rules *allRules = NULL;
	Mine(L, db1, size_L1, 70.0, allRules);
	cout << "\n rules \n" << countRules(allRules);
	print_rules(allRules);

}
