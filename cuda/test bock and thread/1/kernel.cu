#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}


int main(void) {
	int *d_a = 0; 
	int *d_b=0;
	int *d_c=0;
	int size = N * sizeof(int);
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// Cleanup
	//free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}

